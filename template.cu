#include "hip/hip_runtime.h"
#include <gputk.h>

__global__ void vecAdd(float *in1, float *in2, float *out, int len) {
  //@@ Insert code to implement vector addition here
	int currIndex = (blockIdx.x * blockDim.x) + threadIdx.x;
        if(currIndex < len)  {
           out[currIndex] = in1[currIndex] + in2[currIndex];
	}
}

int main(int argc, char **argv) {
  gpuTKArg_t args;
  int inputLength;
  float *hostInput1;
  float *hostInput2;
  float *hostOutput;
  float *deviceInput1;
  float *deviceInput2;
  float *deviceOutput;
  int BUFSZ;

  args = gpuTKArg_read(argc, argv);

  gpuTKTime_start(Generic, "Importing data and creating memory on host");
  hostInput1 =
      (float *)gpuTKImport(gpuTKArg_getInputFile(args, 0), &inputLength);
  hostInput2 =
      (float *)gpuTKImport(gpuTKArg_getInputFile(args, 1), &inputLength);
  hostOutput = (float *)malloc(inputLength * sizeof(float));
  gpuTKTime_stop(Generic, "Importing data and creating memory on host");

  gpuTKLog(TRACE, "The input length is ", inputLength);
  BUFSZ = inputLength * sizeof(float);
  gpuTKTime_start(GPU, "Allocating GPU memory.");
  //@@ Allocate GPU memory here
  hipMalloc((void **) &deviceInput1, BUFSZ);
  hipMalloc((void **) &deviceInput2, BUFSZ);
  hipMalloc((void **) &deviceOutput, BUFSZ);
  gpuTKTime_stop(GPU, "Allocating GPU memory.");

  gpuTKTime_start(GPU, "Copying input memory to the GPU.");
  //@@ Copy memory to the GPU here
  hipMemcpy(deviceInput1, hostInput1, BUFSZ, hipMemcpyHostToDevice);
  hipMemcpy(deviceInput2, hostInput2, BUFSZ, hipMemcpyHostToDevice);

  gpuTKTime_stop(GPU, "Copying input memory to the GPU.");

  //@@ Initialize the grid and block dimensions here
  dim3 threadsPerBlock(256);
  dim3 totalBlocks(inputLength/256 + 1);

  gpuTKTime_start(Compute, "Performing CUDA computation");
  //@@ Launch the GPU Kernel here
  vecAdd<<<totalBlocks, threadsPerBlock>>>(deviceInput1, deviceInput2, deviceOutput, inputLength);

  hipDeviceSynchronize();
  gpuTKTime_stop(Compute, "Performing CUDA computation");

  gpuTKTime_start(Copy, "Copying output memory to the CPU");
  //@@ Copy the GPU memory back to the CPU here
  hipMemcpy(hostOutput,deviceOutput, BUFSZ, hipMemcpyDeviceToHost);

  gpuTKTime_stop(Copy, "Copying output memory to the CPU");

  gpuTKTime_start(GPU, "Freeing GPU Memory");
  //@@ Free the GPU memory here
  hipFree(deviceInput1); hipFree(deviceInput2); hipFree(deviceOutput);
  gpuTKTime_stop(GPU, "Freeing GPU Memory");

  gpuTKSolution(args, hostOutput, inputLength);

  free(hostInput1);
  free(hostInput2);
  free(hostOutput);

  return 0;
}
